#include "hip/hip_runtime.h"
#include "luaT.h"
#include "THC.h"
#include "hip/hip_runtime.h"

#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

// no-overlop
__global__ void output_kernel(float *input, float* output, float* weight, int input_h, int input_w,
                              int output_h, int output_w, int kW, int kH){
   float* ptr_input_plane = input + (blockIdx.x + gridDim.x * blockIdx.y) * input_w * input_h;
   float* ptr_output_plane = output + (blockIdx.x + gridDim.x * blockIdx.y) * output_w * output_h;
   float* weight_plane = weight + blockIdx.x * kW * kH;

   int xout = threadIdx.x;
   int yout = threadIdx.y; 
   const int xout_step = blockDim.x;
   const int yout_step = blockDim.y;
   int xin_start = threadIdx.x * kW;
   int yin_start = threadIdx.y * kH;
   const int xin_step = blockDim.x * kW;
   const int yin_step = blockDim.y * kH;
   int xin_end = (input_w/kW) * kW;  //TODO could this be right?
   int yin_end = (input_h/kH) * kH;

   for (int yin = yin_start; yin < yin_end ; yin += yin_step){
      for (int xin = xin_start; xin < xin_end; xin += xin_step){
         float* ptr_input = ptr_input_plane + xin + yin * input_w;
         float* ptr_output = ptr_output_plane + xout + yout * output_w;
         
         if (xout < output_w && yout < output_h){
            for (int ky = 0; ky < kH && yin + ky < input_h; ky++){
               for (int kx = 0; kx < kW && xin + kx < input_w; kx++){
                  float* weight_plane_elem = weight_plane + kx + ky * kW;
                  float* ptr_input_elem = ptr_input + kx + ky * input_w;
                  *ptr_output += (*ptr_input_elem) * (*weight_plane_elem);
               }   
            }   
         } // end if
         xout += xout_step;
      } // end for xout
      yout += yout_step;
   } // end for yout
}


__global__ void grad_input_kernel(float* input, float* grad_output, float* grad_input, float* weight, int input_h,
                                 int input_w, int output_h, int output_w, int kW, int kH){
   float* ptr_grad_output_plane = grad_output + (blockIdx.x + gridDim.x * blockIdx.y) * output_w * output_h;
   float* ptr_grad_input_plane = grad_input + (blockIdx.x + gridDim.x * blockIdx.y) * input_w * input_h;
   float* weight_plane = weight + blockIdx.x * kW * kH;

    int xout = threadIdx.x;
    int yout = threadIdx.y;
    const int xout_step = blockDim.x;
    const int yout_step = blockDim.y;
    int xin_start = threadIdx.x * kW;
    int yin_start = threadIdx.y * kH;
    const int xin_step = blockDim.x * kW;
    const int yin_step = blockDim.y * kH;
    int xin_end = (input_w/kW) * kW;  //TODO could this be right?
    int yin_end = (input_h/kH) * kH;

   for (int yin = yin_start; yin < yin_end; yin += yin_step){
       for (int xin = xin_start; xin < xin_end; xin += xin_step){
           float* ptr_grad_input = ptr_grad_input_plane + xin + yin * input_w;
           float* ptr_grad_output_elem = ptr_grad_output_plane + xout + yout * output_w;

         if (xout < output_w && yout < output_h){
           for (int ky = 0; ky < kH && yin + ky < input_h; ky++){
            for (int kx = 0; kx < kW && xin + kx < input_w; kx++){
                float* ptr_grad_input_elem = ptr_grad_input + kx + ky * input_w;
                float* weight_plane_elem = weight_plane + kx + ky * kW;
                *ptr_grad_input_elem = (*weight_plane_elem) * (*ptr_grad_output_elem);
            } // end for kx
           } // end for ky
         }
         xout += xout_step;
       } // end for xin
      yout += yout_step;
   } // end for yin
}


__global__ void accGrad_kernel(float* input, float* grad_output, float* grad_weight, float* weight, int input_h,
                               int input_w, int output_h, int output_w, int kW, int kH, float scale){
   float* ptr_input_plane = input + (blockIdx.x + gridDim.x * blockIdx.y) * input_w * input_h;
   float* ptr_grad_output_plane = grad_output + (blockIdx.x + gridDim.x * blockIdx.y) * output_w * output_h;
   float* grad_weight_plane = grad_weight + blockIdx.x * kW * kH;

   int xout = threadIdx.x;
   int yout = threadIdx.y;
   const int xout_step = blockDim.x;
   const int yout_step = blockDim.y;
   int xin_start = threadIdx.x * kW;
   int yin_start = threadIdx.y * kH;
   const int xin_step = blockDim.x * kW;
   const int yin_step = blockDim.y * kH;
   int xin_end = (input_w/kW) * kW;  //TODO could this be right?
   int yin_end = (input_h/kH) * kH;

   for (int yin = yin_start; yin < yin_end; yin += yin_step){
      for (int xin = xin_start; xin < xin_end; xin += xin_step){
         float* ptr_grad_output_plane_elem = ptr_grad_output_plane + xout + yout * output_w;
         float* ptr_input_plane_elem = ptr_input_plane + xin + yin * input_w;
         if (xout < input_w && yout < input_h){
            for (int ky = 0; ky < kH && yin + ky < input_h; ky++){
               for (int kx = 0; kx < kW && xin + kx < input_w; kx++){
                  float* grad_weight_plane_elem = grad_weight_plane + kx + ky * kW;
                  float* ptr_input_plane_elem_elem = ptr_input_plane_elem + kx + ky * input_w;
                  float tmp = scale * (*ptr_input_plane_elem_elem) * (*ptr_grad_output_plane_elem);
                  atomicAdd(grad_weight_plane_elem, tmp);
               }  
            }  
         }
         xout += xout_step;
      } // end for xin   
      yout += yout_step;
   } // end for yin
}


static int cunn_SpatialMlpPooling_updateOutput(lua_State *L){
    THCudaTensor* input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
    THCudaTensor* output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
    THCudaTensor* weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
    int kW = luaT_getfieldcheckint(L, 1, "kW");
    int kH = luaT_getfieldcheckint(L, 1, "kH");
    float* output_data;
    float* input_data;
    float* weight_data;

    long nInputCols = input -> size[3];
    long nInputRows = input -> size[2];
    long nInputPlane = input -> size[1];
    long nBatch = input -> size[0];
    long nOutputCols = nInputCols / kW;
    long nOutputRows = nInputRows / kH;

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(input);

    input_data = THCudaTensor_data(input);
    weight_data = THCudaTensor_data(weight);

    THCudaTensor_resize4d(output, nBatch, nInputPlane, nOutputRows, nOutputCols);
    THCudaTensor_zero(output);

    output_data = THCudaTensor_data(output);

    dim3 blocks(nInputPlane, nBatch);
    dim3 threads(32,8);
    
    output_kernel <<<blocks, threads>>> (input_data, output_data, weight_data, nInputRows, nInputCols, nOutputRows, nOutputCols, kW, kH);

    THCudaTensor_free(input);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        printf("error in SpatialMaxPoolingPos.updateOutput: %s\n", hipGetErrorString(err));
        THError("aborting");
    }
    return 1;
}


static int cunn_SpatialMlpPooling_updateGradInput(lua_State *L){
    THCudaTensor* input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
    THCudaTensor* gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
    THCudaTensor* weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
    THCudaTensor* gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
    int kW = luaT_getfieldcheckint(L, 1, "kW");
    int kH = luaT_getfieldcheckint(L, 1, "kH");

    float* input_data;
    float* weight_data;
    float* gradOutput_data;
    float* gradInput_data;

    long nInputCols = input -> size[3];
    long nInputRows = input -> size[2];
    long nInputPlane = input -> size[1];
    long nBatch = input -> size[0];
    long nOutputCols = nInputCols / kW;
    long nOutputRows = nInputRows / kH;

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    THCudaTensor_resizeAs(gradInput, input);
    THCudaTensor_zero(gradInput);

    input = THCudaTensor_newContiguous(input);
    gradOutput = THCudaTensor_newContiguous(gradOutput);

    input_data = THCudaTensor_data(input);
    gradOutput_data = THCudaTensor_data(gradOutput);
    gradInput_data = THCudaTensor_data(gradInput);
    weight_data = THCudaTensor_data(weight);

    dim3 blocks(nInputPlane, nBatch);
    dim3 threads(32,8);
    
    grad_input_kernel <<<blocks, threads>>> (input_data, gradOutput_data, gradInput_data, weight_data, nInputRows, nInputCols, nOutputRows, nOutputCols, kW, kH);

    THCudaTensor_free(input);
    THCudaTensor_free(gradOutput);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        printf("error in SpatialMaxPoolingPos.updateOutput: %s\n", hipGetErrorString(err));
        THError("aborting");
    }
    return 1;
}

static int cunn_SpatialMlpPooling_accGradParameters(lua_State *L){
    THCudaTensor* input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
    THCudaTensor* gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
    THCudaTensor* weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
    THCudaTensor* gradWeight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
    float scale = luaL_checknumber(L, 4); 
    int kW = luaT_getfieldcheckint(L, 1, "kW");
    int kH = luaT_getfieldcheckint(L, 1, "kH");

    float* input_data;
    float* weight_data;
    float* gradOutput_data;
    float* gradWeight_data;

    long nInputCols = input -> size[3];
    long nInputRows = input -> size[2];
    long nInputPlane = input -> size[1];
    long nBatch = input -> size[0];
    long nOutputCols = nInputCols / kW;
    long nOutputRows = nInputRows / kH;

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    THCudaTensor_resizeAs(gradWeight, input);
    THCudaTensor_zero(gradWeight);

    input = THCudaTensor_newContiguous(input);
    gradOutput = THCudaTensor_newContiguous(gradOutput);

    input_data = THCudaTensor_data(input);
    gradOutput_data = THCudaTensor_data(gradOutput);
    gradWeight_data = THCudaTensor_data(gradWeight);
    weight_data = THCudaTensor_data(weight);

    dim3 blocks(nInputPlane, nBatch);
    dim3 threads(32,8);
    
    accGrad_kernel <<<blocks, threads>>> (input_data, gradOutput_data, gradWeight_data, weight_data, nInputRows, nInputCols, nOutputRows, nOutputCols, kW, kH, scale);

    THCudaTensor_free(input);
    THCudaTensor_free(gradOutput);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        printf("error in SpatialMaxPoolingPos.updateOutput: %s\n", hipGetErrorString(err));
        THError("aborting");
    }
    return 1;
}

static const struct luaL_Reg cunn_SpatialMlpPooling__ [] = {
   {"SpatialMlpPooling_updateOutput", cunn_SpatialMlpPooling_updateOutput},
   {"SpatialMlpPooling_updateGradInput", cunn_SpatialMlpPooling_updateGradInput},
   {"SpatialMlpPooling_accGradParameters", cunn_SpatialMlpPooling_accGradParameters},
   {NULL, NULL}
};

void cunn_SpatialMlpPooling_init(lua_State* L){
   luaL_openlib(L, "jz", cunn_SpatialMlpPooling__, 0) ;
}
