#include "hip/hip_runtime.h"
#include "luaT.h"
#include "THC.h"
#include "hip/hip_runtime.h"

#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

//no-overlap
__global__ void output_kernel(float *input, float* output, )
